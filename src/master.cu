/*
 * MicroHH
 * Copyright (c) 2011-2017 Chiel van Heerwaarden
 * Copyright (c) 2011-2017 Thijs Heus
 * Copyright (c) 2014-2017 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cstdarg>
#include <cstdio>
#include <iostream>
#include <sstream>
#include <hip/hip_runtime_api.h>
#include "master.h"
#include "tools.h"
#define ENV_LOCAL_RANK	"MV2_COMM_WORLD_LOCAL_RANK"

void Master::SetDeviceBeforeInit() // It's tested with single node multi-GPU.
{
  #ifdef USEMPI
	int devCount = 0;
  cuda_safe_call(hipGetDeviceCount(&devCount));
  cuda_safe_call(hipSetDevice(md.mpiid % devCount));
  cuda_check_error();
  #endif
}
